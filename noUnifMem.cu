
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

typedef struct
{
  int size;
  char *_string;
} string_t;

__global__ void string_append(string_t*, string_t*, string_t*);

int main(void)
{
  int size;

  string_t *str1_host = (string_t *)malloc(sizeof(string_t));
  char _string1[] = "Hello, ";
  size = sizeof(_string1);
  str1_host->size = size;
  str1_host->_string = (char *)malloc(size * sizeof(char));
  memcpy(str1_host->_string, _string1, size);


  string_t *str2_host = (string_t *)malloc(sizeof(string_t));
  char _string2[] = "world!\n";
  size = sizeof(_string2);
  str1_host->size = size;
  str1_host->_string = (char *)malloc(size * sizeof(char));
  memcpy(str1_host->_string, _string2, size);

  string_t *result_host = (string_t *)malloc(sizeof(string_t)); 
  result_host->size = str1_host->size+str2_host->size;
  result_host->_string = (char *)malloc(result_host->size * result_host->size);
   
  string_t *str1_device;
  
  string_t *str2_device;
  string_t *result_device;
  

  // allocate memory on global memory (device).
  hipMalloc(&str1_device, sizeof(string_t));
  char *string1_device;
  hipMalloc(&string1_device, str1_host->size);
  //cudaMalloc(&(str1_device->_string), str1_host->size)  

  hipMalloc(&str2_device, sizeof(string_t));
  char *string2_device;
  hipMalloc(&string2_device, str2_host->size);
  //cudaMalloc(&(str2_device->_string), str2_host->size);

  hipMalloc(&result_device, sizeof(string_t));
  char *res_string_device;
  hipMalloc(&res_string_device, result_host->size);
  hipMemcpy(&(result_device->_string), &res_string_device, sizeof(char *),
  hipMemcpyDeviceToDevice);
  //cudaMalloc(&(result_device->_string), result_host->size);

  // copy data (str1_host and str2_host) to 
  // global memory (device). 
  hipMemcpy(str1_device, str1_host, sizeof(string_t), hipMemcpyHostToDevice);
  hipMemcpy(string1_device, str1_host->_string, str1_host->size, 
  hipMemcpyHostToDevice);
  hipMemcpy(&(str1_device->_string), &string1_device, sizeof(char *), 
  hipMemcpyHostToDevice);

  //cudaMemcpy(str1_device->_string, str1_host->_string, str1_host->size,
  //cudaMemcpyHostToDevice);
  
  hipMemcpy(str2_device, str2_host, sizeof(string_t), hipMemcpyHostToDevice);
  hipMemcpy(string2_device, str2_host->_string, str2_host->size, 
  hipMemcpyHostToDevice);
  hipMemcpy(&(str2_device->_string), &string2_device, sizeof(char *), 
  hipMemcpyHostToDevice);

  //cudaMemcpy(str2_device, str2_host, str2_host->size, 
  //cudaMemcpyHostToDevice);

  // append str1 and str2 in result (it's done on the device)
  string_append<<<1,1>>>(str1_device, str2_device, result_device);
  
  // copy the result back to the CPU memory
  hipMemcpy(result_host->_string, result_device->_string, result_host->size, 
  hipMemcpyDeviceToHost);

  // check the result on the CPU. 
  // It should print "Hello, world!."
  printf("%s", result_host->_string);  


  exit(EXIT_SUCCESS);
}

__global__ void string_append(string_t* str1, string_t* str2, string_t *str3)
{
  memcpy(str3->_string, str1->_string, str1->size);
  memcpy(str3->_string+(str1->size), str2->_string, str2->size);
}

