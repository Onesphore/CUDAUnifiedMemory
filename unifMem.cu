
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// a struct 
typedef struct
{
  int size;
  char *_string;
} string_t;

__global__ void string_append(string_t*, string_t*, string_t*);

int main(void)
{
  string_t *str1, *str2, *str3;
  size_t size1 = strlen("Hello, ");
  size_t size2 = sizeof("world!");
  
  // allocate memory for str1, str2, and str3 
  // and for their string members.
  hipMallocManaged(&str1, sizeof(string_t));
  hipMallocManaged(&(str1->_string), size1*sizeof(char));

  hipMallocManaged(&str2, sizeof(string_t));
  hipMallocManaged(&(str2->_string), size2*sizeof(char));

  hipMallocManaged(&str3, sizeof(string_t));
  hipMallocManaged(&(str3->_string), (size1+size2) * sizeof(char));
  
  // the CPU and the GPU can directly access memory allocated to str's 
  
  //In the following 4 lines the Host is accessing
  // the memory allocated via cudaMallocManaged().
  str1->size = size1;
  memcpy(str1->_string, "Hello, ", size1);

  str2->size = size2;
  memcpy(str2->_string, "world!", size2);
  
  // Since the kernel has been declared/defined with 
  // "__global__" keyword we are sure it will run on the device.
  // So the device will be accessing memory allocated via cudaMallocManaged().
  string_append<<<1, 1>>>(str1, str2, str3);

  // the device and the host should be synchronized
  // before we call printf in the host.
  hipDeviceSynchronize();
  
  printf("%s\n", str3->_string);

  exit(EXIT_SUCCESS);
}



// a kernel to append 2 strings in a 3rd string.
__global__ void string_append(string_t *s1, string_t *s2, string_t *s3)
{
  memcpy(s3->_string, s1->_string, s1->size);
  memcpy(s3->_string+(s1->size), s2->_string, s2->size);
}
